#include "hip/hip_runtime.h"
#define _POSIX_C_SOURCE 200809L

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "libcxl.h"

#include "defines.h"
#include "batch.h"
#include "utils.h"
#include "psl.h"
#include "batch.c"
#include "utils.c"


#define BILLION 1000000000L




//****************codes below added by shanshan*************//

typedef struct struct_NUM_ADD
  {
    short read_number;
    short haplotype_number;
    int address_array;
  } NUM_ADD;

typedef struct struct_parameters{
  float distm_simi[32];
  float distm_diff[32];
  float alpha[32];
  float beta[32];
  float delta[32];
  float upsilon[32];
  float eta[32];
  float zeta[32];
} t_parameters;



__global__ void  pairHMM( int size, char * data,NUM_ADD * num_add, float * result) // what is the maximum number of parameters?
  {
   int offset=blockIdx.x;

   while(offset<size)
   {  
    //as each time it will deal with 2 read&haplotype pairs
    // each block deal with one pairs of haplotype & read
    NUM_ADD number_address;
    number_address=num_add[offset];

    int read_number=number_address.read_number; 
    int haplotype_number=number_address.haplotype_number;
   
    char * read_base_array=(char *)(data+number_address.address_array); // to caculate the address of read_base_array. 
    char4 * haplotype_base_array=(char4 * )(read_base_array+(read_number+127)/128*128);
    int aa=(haplotype_number+3)/4;
    t_parameters *parameter_array=(t_parameters *) (read_base_array+(read_number+127)/128*128+ (aa*4+127)/128*128);
      

    __shared__ char haplotype_base_in_char[350];
    int hh=(haplotype_number+4-1)/4;
    int tt=(hh+blockDim.x-1)/blockDim.x;
    for(int ii=0;ii<tt;ii++)
    { 
      int aa=threadIdx.x+ii*blockDim.x;
      if(aa< hh)
      {
      char4 haplotype_base_in_thread;
      haplotype_base_in_thread=haplotype_base_array[aa]; //Is it right to get data from global memory
      haplotype_base_in_char[aa*4]=haplotype_base_in_thread.x;
      haplotype_base_in_char[aa*4+1]=haplotype_base_in_thread.y;
      haplotype_base_in_char[aa*4+2]=haplotype_base_in_thread.z;
      haplotype_base_in_char[aa*4+3]=haplotype_base_in_thread.w;
}
    }
    __syncthreads();

    float MM, DD,II;
    float Qm,Qm_1,alpha,beta,delta,epsion,xiksi,thet;
    float D_0=(ldexpf(1.f, 120))/(float)haplotype_number;
    
    __shared__ float MM_stored[270];// as long as the haplotype
    __shared__ float DD_stored[270];
    __shared__ float II_stored[270];
    float result_block=0;
    int round=(read_number+blockDim.x-1)/blockDim.x;
      
    int round_size;
    char read_base;
    for(int i=0;i<round;i++)
    {
      round_size=(read_number>blockDim.x)?blockDim.x: read_number;
      read_number=(read_number>blockDim.x)?read_number-blockDim.x:0; // read_num is the remaining length at this round
      
      if(threadIdx.x<round_size ) // tid is from 0 ~ round_size-1
      {
        read_base=read_base_array[threadIdx.x+blockDim.x*i];
       
        Qm_1=parameter_array[i].distm_simi[threadIdx.x];
        Qm=parameter_array[i].distm_diff[threadIdx.x];
        alpha=parameter_array[i].alpha[threadIdx.x];
        beta=parameter_array[i].beta[threadIdx.x];
        delta=parameter_array[i].delta[threadIdx.x];
        epsion=parameter_array[i].upsilon[threadIdx.x];
        xiksi=parameter_array[i].eta[threadIdx.x];
        thet=parameter_array[i].zeta[threadIdx.x];


 }
        
      float M=0; //now 
      float I=0; //now
      float D=0; //now
      float MMM=0;//up left
      float DDD=0;//up left
      float III=0;//up left
      if(threadIdx.x==0&&i==0) DDD=D_0; // Just in the first round, it need to be D_0
      
      int current_haplotype_id=0;
      for(int j=0;j<round_size+haplotype_number-1;j++)
      { 
        int aa=j-threadIdx.x; 
        if( aa>=0 && (current_haplotype_id<haplotype_number))
        {
          if(threadIdx.x==0) // if it is the second or more round
                                        {
                                                if(i>0)
                                                {
                                                MM=MM_stored[current_haplotype_id];
                                                II=II_stored[current_haplotype_id];
                                                DD=DD_stored[current_haplotype_id];
                                                }
                                                else
                                                {
                                                MM=0;
                                                II=0;
                                                DD=D_0;
                                                }

                                        }   
                                        float MID=__fadd_rn(III,DDD);
                                        DDD=DD;
                                        III=II;
                                        float DDM=__fmul_rn(M,xiksi);
                                        float IIMI=__fmul_rn(II,epsion);
                                        float MIIDD=__fmul_rn(beta,MID);
                                        char haplotype_base_each=haplotype_base_in_char[current_haplotype_id];
                                        float aa=(haplotype_base_each==read_base)? Qm_1:Qm;
                                        D=__fmaf_rn(D,thet,DDM);
                                        I=__fmaf_rn(MM,delta,IIMI);
                                        float MMID=__fmaf_rn(alpha,MMM,MIIDD);
                                        MMM=MM;
                                        current_haplotype_id++;
                                        M=__fmul_rn(aa,MMID);
                                        II=I;
                                        DD=D;
                                        MM=M;


                                  }
        
                                if(threadIdx.x==round_size-1 && i<round-1) // tid is the last thread but there are more round
                                {
                                        MM_stored[current_haplotype_id-1]=M;
                                        II_stored[current_haplotype_id-1]=I;
                                        DD_stored[current_haplotype_id-1]=D;
                                }
                                if(threadIdx.x==round_size-1 && i==round-1)
                                        result_block=__fadd_rn(result_block,__fadd_rn(M,I));

                                MM=__shfl_up(MM,1);
                                II=__shfl_up(II,1);
                                DD=__shfl_up(DD,1);
      }
    }
    if(threadIdx.x==round_size-1) 
    {
      result[offset]=result_block;
    } 
    offset+=gridDim.x;  
   }

}
//****************above codes added by shanshan*************//



int main (int argc, char *argv[]) {
  struct timespec hwstart, hwend;
  //struct cxl_afu_h *afu;
  void             *batch;
  t_result         *result_hw;
  t_result         *result_sw;
  t_workload       *workload;
  t_batch          *batches;  
  
  unsigned char    show_table = 0;
  unsigned char		 show_results = 0;
  unsigned char 	 calculate_sw = 0;
  double					 clock_sw;
  double					 clock_hw;
  
  uint64_t   			 threads = 1;

  DEBUG_PRINT("Parsing input arguments...\n");
  if (argc < 5) {
    fprintf(stderr, "ERROR: Correct usage is: %s <-f = file, -m = manual> ... \n-m: <pairs> <X> <Y> ... \n-f: <input file>\n... <number of threads*> <sw solve?*> <show results?*> <show MID table?*> (* is optional)\n", APP_NAME);
    return -1;
  } 
  else {
		if (strncmp(argv[1],"-f",2)==0) {
			if ((workload = load_workload(argv[2])) == NULL) {
				fprintf(stderr, "ERROR: %s cannot be opened.\n", argv[2]);
				return -1;
			}
			if (argc >= 4) threads			= strtoul(argv[3], NULL, 0);
			if (argc >= 5) calculate_sw = strtoul(argv[4], NULL, 0);
			if (argc >= 6) show_results = strtoul(argv[5], NULL, 0);
			if (argc >= 7) show_table   = strtoul(argv[6], NULL, 0);
			
			if (threads <= 0) threads = omp_get_max_threads();
			
			BENCH_PRINT("%s, ", argv[2]);
			BENCH_PRINT("%8d, ", (int) workload->pairs);
			BENCH_PRINT("%8d, ", (int) threads);
		}
		else if (strncmp(argv[1],"-m",2)==0) {
      DEBUG_PRINT("Manual input mode selected. %d arguments supplied.\n", argc);
			int pairs = strtoul(argv[2], NULL, 0);
			int x = strtoul(argv[3], NULL, 0);
			int y = strtoul(argv[4], NULL, 0);
      
      workload = gen_workload(pairs, x, y);
      
			if (argc >= 6) threads			= strtoul(argv[5], NULL, 0);
			if (argc >= 7) calculate_sw = strtoul(argv[6], NULL, 0);
			if (argc >= 8) show_results = strtoul(argv[7], NULL, 0);
			if (argc >= 9) show_table   = strtoul(argv[8], NULL, 0);
			if (threads <= 0) threads = omp_get_max_threads();

			BENCH_PRINT("M, ");
			BENCH_PRINT("%8d, %8d, %8d, ", workload->pairs, x, y);
			BENCH_PRINT("%8d, ", (int) threads);
		}
		else
		{
			fprintf(stderr, "ERROR: Correct usage is: %s <-f = file, -m = manual> ... \n-m: <pairs> <X> <Y> ... \n-f: <input file>\n... <number of threads*> <sw solve?*> <show results?*> <show MID table?*> (* is optional)\n", APP_NAME);
			return EXIT_FAILURE;
		}
  }
  
  BENCH_PRINT("%16lu, ",workload->cups_req);
  
  DEBUG_PRINT("Total workload bytes: %17d \n", (unsigned int) workload->bytes);
  DEBUG_PRINT("CUPS required       : %17lu \n", workload->cups_req);     
  DEBUG_PRINT("Allocating memory for %d batches and %d results...\n", (unsigned int) workload->batches, (unsigned int) workload->pairs);

  if (posix_memalign( (void **) &batch, CACHELINE_BYTES, workload->bytes)) {
    perror("Could not allocate memory to store the batches.\n");
    return -1;
  }

  if (posix_memalign( (void **) &result_hw, CACHELINE_BYTES, sizeof(t_result) * workload->batches * PIPE_DEPTH)) {
    perror("Could not allocate memory to store hardware results.\n");
    return -1;
  }

  if (posix_memalign( (void **) &result_sw, CACHELINE_BYTES, sizeof(t_result) * workload->batches * PIPE_DEPTH)) {
    perror("Could not allocate memory to store software results.\n");
    return -1;
  }

  DEBUG_PRINT("Clearing batch and host result memory ...\n");
  memset(result_sw, 0xFF, sizeof(t_result)  * workload->batches * PIPE_DEPTH);
  memset(batch,     0x00, workload->bytes);



  DEBUG_PRINT("Filling batches...\n");

  clock_sw = omp_get_wtime();
  
  void * batch_cur = batch;  
  
  batches = (t_batch*) malloc(sizeof(t_batch) * workload->batches);
      
  for (int q = 0; q < workload->batches; q++) {
    init_batch_address(&batches[q], batch_cur, workload->bx[q], workload->by[q]);
    fill_batch(&batches[q], workload->bx[q], workload->by[q], 1.0);
    print_batch_info(&batches[q]);    
    batch_cur = (void*) ((uint64_t) batch_cur + (uint64_t) workload->bbytes[q]);
  }





  clock_sw = omp_get_wtime() - clock_sw;
  BENCH_PRINT("%16f,",clock_sw);  
  
  DEBUG_PRINT("Calculating on host...\n");

 // printf("\n software start \n") ;
  clock_sw = omp_get_wtime();
  
  //print_batch_memory(batch, workload->bbytes[0] + workload->bbytes[1]);
    
  if (calculate_sw) 
  {
	omp_set_num_threads(threads);
	#pragma omp parallel for
     for (int q = 0; q < workload->batches; q++) {
      
      int x = workload->bx[q];
      int y = workload->by[q];
	 float * M = (float*)malloc(sizeof(float) * (y+1) * (x+1));
	 float * I = (float*)malloc(sizeof(float) * (y+1) * (x+1));
	 float * D = (float*)malloc(sizeof(float) * (y+1) * (x+1));
		       
      // Calculate results
	for (int p = 0; p < PIPE_DEPTH; p++) 
	{
		    calculate_mids(&batches[q], p, x, y, M, I, D);
        
        result_sw[q*PIPE_DEPTH+p].values[0] = 0.0;
        for (int c = 0; c < y+1; c++)
        {
       // WARNING: THIS IS BECAUSE FLOATING POINT ADDITION IS NOT ASSOCIATIVE
          result_sw[q*PIPE_DEPTH+p].values[0] += M[(y+1)*x+c];
          result_sw[q*PIPE_DEPTH+p].values[0] += I[(y+1)*x+c];
      }
	//printf("software result    %e\n", result_sw[q*PIPE_DEPTH+p].values[0]);
	 if (show_table != 0) {
		      print_mid_table(&batches[q], p, x, y, M, I, D);
          fflush(stdout);
		    }
		  }
		  
		  free(M);
		  free(I);
		  free(D);
		}
  }

  clock_sw = omp_get_wtime() - clock_sw;
  
  if (calculate_sw) {
		BENCH_PRINT("%16f, ", clock_sw);
    BENCH_PRINT("%16f, ", workload->cups_req / clock_sw / 1000000);
  }
	else
  {
    BENCH_PRINT("%16f,",0.0);
    BENCH_PRINT("%16f,",0.0);
  }
    
  
  
  DEBUG_PRINT("%d %d\n",calculate_sw, show_results);
  
  if (calculate_sw && (show_results > 0)) {
  	print_results(result_sw, workload->batches);
  }
  
  DEBUG_PRINT("Clearing result memory\n");
  memset(result_hw, 0xFF, sizeof(t_result)  * workload->batches * PIPE_DEPTH);
  
  //printf("\nSoftware end\n") ;
 // DEBUG_PRINT("Opening device: %s ...\n", DEVICE);
 // afu = cxl_afu_open_dev ((char*) (DEVICE));
 // if (!afu) { perror ("cxl_afu_open_dev"); return -1; }




 hipSetDevice(0);
 //start GPU programming
  
  //change data format
  //**********in each batch, there is only one pair of read and haplotype. I change the value of PIPE_DEPTH in define.h file
  //memory on host
  struct timespec start,finish;
 
    double computation_time=0;
  int size=workload->batches; // how many pairs in the workloads 
  char * data_h_total;
  data_h_total=(char*) malloc(size*10000*sizeof(char)+sizeof(NUM_ADD)*size);

  NUM_ADD * data_num_add=(NUM_ADD *) (data_h_total);
  char * data_h=data_h_total+(size*sizeof(NUM_ADD)+127)/128*128;  // to make sure the address is aligned

  //memory on GPU
  char * result_d_total;
  hipMalloc( (char **) &result_d_total, size*10000*sizeof(char)+(size*sizeof(NUM_ADD)+127)/128*128+(size*sizeof(float)+127)/128*128);
  char * data_d_total=result_d_total+(size*sizeof(float)+127)/128*128; // to make sure the address is aligned.


  int data_size=0;
  //for each pair
  for(int q=0;q<workload->batches;q++)
  {
   
    int read_size_new=workload->bx[q];
    int haplotype_size_new=workload->by[q];
    //change read
    char read_base_new[500];
    for(int i=0;i<read_size_new;i++)
    {
      read_base_new[i]=batches[q].read[i].base[0];

   }
    //change haplotype
    int haplotype_size_new_new=(haplotype_size_new+3)/4;
    char4 haplotype_base_new[150];
    for(int i=0;i<haplotype_size_new_new;i++)
    {
          haplotype_base_new[i].x=batches[q].hapl[i*4].base[0];
    	 if(i*4+1<haplotype_size_new)
	{
          haplotype_base_new[i].y=batches[q].hapl[i*4+1].base[0];
         }
	 if(i*4+2<haplotype_size_new)
        {
	  haplotype_base_new[i].z=batches[q].hapl[i*4+2].base[0];
        }
	  if(i*4+3<haplotype_size_new)
        {  haplotype_base_new[i].w=batches[q].hapl[i*4+3].base[0];
   	 }	
    }


    //change parameter
    t_parameters pa[20];
    int aa=(read_size_new+31)/32;
    for(int i=0;i<aa;i++)
    {
      for(int j=0;j<32;j++)
      {
        if(i*32+j<read_size_new)
        {
        pa[i].distm_simi[j]=batches[q].prob[i*32+j].p[7].f;
        pa[i].distm_diff[j]=batches[q].prob[i*32+j].p[6].f;
        pa[i].alpha[j]=batches[q].prob[i*32+j].p[5].f;
        pa[i].beta[j]=batches[q].prob[i*32+j].p[4].f;
        pa[i].delta[j]=batches[q].prob[i*32+j].p[3].f;
        pa[i].upsilon[j]=batches[q].prob[i*32+j].p[2].f;
        pa[i].eta[j]=batches[q].prob[i*32+j].p[1].f;
        pa[i].zeta[j]=batches[q].prob[i*32+j].p[0].f;


	    }
      }
    }

        data_num_add[q].read_number=read_size_new;
        data_num_add[q].haplotype_number=haplotype_size_new;
        data_num_add[q].address_array=data_size;

        memcpy(data_h,read_base_new,sizeof(char)*read_size_new);
        
        data_h+=(read_size_new+127)/128*128;
        data_size+=(read_size_new+127)/128*128;

        memcpy(data_h,haplotype_base_new,sizeof(char4)* haplotype_size_new_new);
        data_h+=(haplotype_size_new_new*sizeof(char4)+127)/128*128;
        data_size+=(haplotype_size_new_new*sizeof(char4)+127)/128*128;
      
        memcpy(data_h,pa,sizeof(t_parameters) *aa);
        data_h+=sizeof(t_parameters)*aa;
        data_size+=sizeof(t_parameters)*aa;
      
  }

      int data_size_to_copy=data_size+(size*sizeof(NUM_ADD)+127)/128*128;
      float * result_h=(float *) malloc(sizeof(float)*size);
      
      clock_hw = omp_get_wtime();
      clock_gettime(CLOCK_MONOTONIC_RAW, &hwstart); /* mark start time */
     
      hipMemcpy(data_d_total,data_h_total,data_size_to_copy,hipMemcpyHostToDevice);
      NUM_ADD * num_add_d=(NUM_ADD *) (data_d_total);
      char * data_d=data_d_total+(sizeof(NUM_ADD)*size+127)/128*128;
      

     //  call kernel
      dim3 block(32);
      dim3 grid(size);
         pairHMM<<<grid,block>>> (size,data_d, num_add_d,(float *)result_d_total);

     hipMemcpy(result_h,result_d_total,size*sizeof(float),hipMemcpyDeviceToHost);

     // for(int i=0;i<size;i++)
      //   printf("GPU result   i=%d  %e\n",i, result_h[i]);
      //hipDeviceReset();
    clock_gettime(CLOCK_MONOTONIC_RAW, &hwend); 
    clock_hw =omp_get_wtime() - clock_hw;
    

    uint64_t diff = BILLION * (hwend.tv_sec - hwstart.tv_sec) + hwend.tv_nsec - hwstart.tv_nsec; 
     free(result_h);
     free(data_h_total);
     hipFree(result_d_total);

  int errs = 0;
  
  if (calculate_sw)
  {
	  errs = count_errors((uint32_t *)result_hw,(uint32_t *)result_sw,workload->batches);
	}
  
  DEBUG_PRINT("Errors: %d\n",errs);

   
     BENCH_PRINT(" %16f,",clock_hw);
     BENCH_PRINT("  %16llu,", (long long unsigned int) diff); 
  
      BENCH_PRINT(" %16f,", ((double)workload->cups_req / (double)clock_hw) / 1000000);
    
	if (calculate_sw)
	{
	BENCH_PRINT("%16f,",clock_sw / clock_hw);
	}
	else
		BENCH_PRINT(" %16f,",0.0);
	
	BENCH_PRINT("%16d",errs);


	BENCH_PRINT("\n");
	free(workload);
	free(result_sw);
  	free(batch);

  return 0;
}

